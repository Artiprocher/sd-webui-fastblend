
#include <hip/hip_runtime.h>
extern "C"
__global__ void remap(const int height, const int width, const int channel,
                      const int patch_size, const int pad_size,
                      const float *source_style, const int *nnf, float *target_style) {
  const int r = (patch_size - 1) / 2;
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  if (x >= height or y >= width)
    return;
  const int z =
      blockIdx.z * (height + pad_size * 2) * (width + pad_size * 2) * channel;
  const int pid = (x + pad_size) * (width + pad_size * 2) + (y + pad_size);
  const int min_px = x < r ? -x : -r;
  const int max_px = x + r > height - 1 ? height - 1 - x : r;
  const int min_py = y < r ? -y : -r;
  const int max_py = y + r > width - 1 ? width - 1 - y : r;
  int num = 0;
  int imsize = height * width * 2;
  for (int px = min_px; px <= max_px; px++) {
    for (int py = min_py; py <= max_py; py++) {
      const int nid = (x + px) * width + y + py;
      const int x_ = nnf[blockIdx.z * imsize + nid * 2 + 0] - px;
      const int y_ = nnf[blockIdx.z * imsize + nid * 2 + 1] - py;
      if (x_ < 0 or y_ < 0 or x_ >= height or y_ >= width)
        continue;
      const int pid_ =
          (x_ + pad_size) * (width + pad_size * 2) + (y_ + pad_size);
      num++;
      for (int c = 0; c < channel; c++) {
        target_style[z + pid * channel + c] +=
            source_style[z + pid_ * channel + c];
      }
    }
  }
  for (int c = 0; c < channel; c++) {
    target_style[z + pid * channel + c] /= num;
  }
}
